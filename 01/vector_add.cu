
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;
  
  for(int i = idx; i < N; i = i+stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  int threads_per_block = 1024;
  int num_blocks = (N + threads_per_block - 1)/threads_per_block;
  

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);
 
  addVectorsInto<<<num_blocks, threads_per_block>>>(c, a, b, N);
  
  checkCuda(hipDeviceSynchronize());
  

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
